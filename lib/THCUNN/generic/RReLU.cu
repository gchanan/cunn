
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/RReLU.cu"
#else

#include "../common.h"

void THNN_(RReLU_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *noise,
           double lower,
           double upper,
           bool train,
           bool inplace,
           void *generator)
{
  THCUNN_assertSameGPU_generic(state, 3, input, output, noise);
  struct curandStateMtgp32* gen_states = THCRandom_generatorStates(state);

  if (train)
  {
    input = THCTensor_(newContiguous)(state, input);
    THCTensor_(resizeAs)(state, noise, input);
    real *input_data = THCTensor_(data)(state, input);
    real *noise_data = THCTensor_(data)(state, noise);
    long n = THCTensor_(nElement)(state, input);
    if (inplace)
    {
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, gen_states, input_data, noise_data, input_data, lower, upper);
      THCTensor_(set)(state, output, input);
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      real *output_data = THCTensor_(data)(state, output);
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, gen_states, input_data, noise_data, output_data, lower, upper);
    }
    THCudaCheck(cudaGetLastError());
    THCTensor_(free)(state, input);
  }
  else
  {
    const real negSlope = ScalarConvert<double, real>::to((lower + upper) / 2);
    if (inplace)
    {
      THC_pointwiseApply1(state, input, RReLUUpdateOutputEvalIP_functor<real>(negSlope));
      THCTensor_(set)(state, output, input);
    }
    else
    {
      THCTensor_(resizeAs)(state, output, input);
      THC_pointwiseApply2(state, output, input, RReLUUpdateOutputEval_functor<real>(negSlope));
    }
  }
}

void THNN_(RReLU_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *noise,
           double lower,
           double upper,
           bool train,
           bool inplace)
{
  THCUNN_assertSameGPU_generic(state, 4, input, gradOutput, gradInput, noise);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  if (train && upper - lower > 1E-6)    // e.g. if upper == lower, RReLU behaves like LeakyReLU
  {
    // multiply the gradient by the noise tensor
    if (inplace)
    {
      THCTensor_(cmul)(state, gradOutput, gradOutput, noise);
      THCTensor_(set)(state, gradInput, gradOutput);
    }
    else
    {
      THCTensor_(resizeAs)(state, gradInput, input);
      THCTensor_(cmul)(state, gradInput, gradOutput, noise);
    }
  }
  else
  {
    // use constant factor for negative input values
    const real negSlope = ScalarConvert<double, real>::to((lower + upper) / 2);
    if (inplace)
    {
      THC_pointwiseApply2(state, gradOutput, input, RReLUupdateGradInputEvalIP_functor<real>(negSlope));
      THCTensor_(set)(state, gradInput, gradOutput);
    }
    else
    {
      THCTensor_(resizeAs)(state, gradInput, input);
      THC_pointwiseApply3(state, gradInput, gradOutput, input, RReLUupdateGradInputEval_functor<real>(negSlope));
    }
  }

  THCTensor_(free)(state, gradOutput);
}

#endif