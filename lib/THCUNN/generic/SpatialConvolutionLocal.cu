
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialConvolutionLocal.cu"
#else

void THNN_(SpatialConvolutionLocal_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *weight,
           THCTensor *bias,
           THCTensor *finput,
           THCTensor *fgradInput,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           long inputWidth, long inputHeight,
           long outputWidth, long outputHeight)
{
  THCUNN_assertSameGPU_generic(state, 5, input, output, weight,
                                 bias, finput);

  long nInputPlane = THCTensor_(size)(state,weight,2)/(kW*kH);
  long nOutputPlane = THCTensor_(size)(state,weight,1);

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, nInputPlane, inputHeight, inputWidth);
  }

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize4d)(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Augment the input
  THCTensor_(resize3d)(state, finput, batchSize, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *finput_n = THCTensor_(new)(state);
  THCTensor *output_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    THCTensor *finput3d, *output3d;
    THCTensor *wslice = THCTensor_(new)(state);
    THCTensor *islice = THCTensor_(new)(state);
    THCTensor *oslice = THCTensor_(new)(state);

    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, finput_n, finput, 0, elt);
    THCTensor_(select)(state, output_n, output, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, finput_n)
    );

    output3d = THCTensor_(newWithStorage3d)(state, output_n->storage, output_n->storageOffset,
                                             outputHeight*outputWidth, 1,
                                             nOutputPlane, outputHeight*outputWidth,
                                             1, nOutputPlane*outputHeight*outputWidth);

    finput3d = THCTensor_(newWithStorage3d)(state, finput_n->storage, finput_n->storageOffset,
                                             outputHeight*outputWidth, 1,
                                             kW*kH*nInputPlane, outputHeight*outputWidth,
                                             1, kW*kH*nInputPlane*outputHeight*outputWidth);

    THCTensor_(copy)(state, output_n, bias);

    for (int i = 0; i < outputHeight; i++) {
      for(int j = 0; j < outputWidth; j++) {
        int sliceidx = i * outputWidth + j;
        THCTensor_(select)(state, wslice, weight, 0, sliceidx);
        THCTensor_(select)(state, islice, finput3d, 0, sliceidx);
        THCTensor_(select)(state, oslice, output3d, 0, sliceidx);
        THCTensor_(addmm)(state, oslice, ScalarConvert<int, real>::to(1), oslice, ScalarConvert<int, real>::to(1), wslice, islice);
      }
    }


    // weight:    oH*oW x nOutputPlane x nInputPlane*kH*kW
    // finput3d:  oH*oW x nInputPlane*kH*kW x 1
    // THCTensor_(baddbmm)(state, output3d, 1.0, output3d, 1.0, weight, finput3d);
    // output3d:  oH*oW x nOutputPlane x 1

    THCTensor_(free)(state, output3d);
    THCTensor_(free)(state, finput3d);
    THCTensor_(free)(state, wslice);
    THCTensor_(free)(state, islice);
    THCTensor_(free)(state, oslice);
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, finput_n);
  THCTensor_(free)(state, output_n);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize3d)(state, output, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }
}

void THNN_(SpatialConvolutionLocal_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *weight,
           THCTensor *finput,
           THCTensor *fgradInput,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           long inputWidth, long inputHeight,
           long outputWidth, long outputHeight)
{
  THCUNN_assertSameGPU_generic(state, 5, input, gradOutput, weight,
                                 fgradInput, gradInput);

  long nInputPlane = THCTensor_(size)(state,weight,2)/(kW*kH);
  long nOutputPlane = THCTensor_(size)(state,weight,1);

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize4d)(state, gradOutput, 1, nOutputPlane, outputHeight, outputWidth);
  }

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCTensor_(resize4d)(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCTensor_(resize3d)(state, fgradInput, batchSize, nInputPlane*kW*kH, outputHeight*outputWidth);

  // Helpers
  THCTensor *gradInput_n = THCTensor_(new)(state);
  THCTensor *fgradInput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  THCTensor_(transpose)(state, weight, weight, 1, 2);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    THCTensor *gradOutput3d, *fgradInput3d;
    THCTensor *wslice = THCTensor_(new)(state);
    THCTensor *gislice = THCTensor_(new)(state);
    THCTensor *goslice = THCTensor_(new)(state);

    // Matrix mulitply per sample:
    THCTensor_(select)(state, gradInput_n, gradInput, 0, elt);
    THCTensor_(select)(state, fgradInput_n, fgradInput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    gradOutput3d = THCTensor_(newWithStorage3d)(state, gradOutput_n->storage, gradOutput_n->storageOffset,
                                               outputHeight*outputWidth, 1,
                                               nOutputPlane, outputHeight*outputWidth,
                                               1, nOutputPlane*outputHeight*outputWidth);
    fgradInput3d = THCTensor_(newWithStorage3d)(state, fgradInput_n->storage, fgradInput_n->storageOffset,
                                               outputHeight*outputWidth, 1,
                                               kW*kH*nInputPlane, outputHeight*outputWidth,
                                               1, kW*kH*nInputPlane*outputHeight*outputWidth);

    for (int i = 0; i < outputHeight; i++) {
      for(int j = 0; j < outputWidth; j++) {
        int sliceidx = i * outputWidth + j;
        THCTensor_(select)(state, wslice, weight, 0, sliceidx);
        THCTensor_(select)(state, gislice, fgradInput3d, 0, sliceidx);
        THCTensor_(select)(state, goslice, gradOutput3d, 0, sliceidx);
        THCTensor_(addmm)(state, gislice, ScalarConvert<int, real>::to(0), gislice, ScalarConvert<int, real>::to(1), wslice, goslice);
      }
    }

    // weight:        oH*oW x nInputPlane*kH*kW x nOutputPlane
    // gradOutput3d:  oH*oW x nOutputPlane x 1
    //THCTensor_(baddbmm)(state, fgradInput3d, 0.0, fgradInput3d, 1.0, weight, gradOutput3d);
    // fgradInput3d:  oH*oW x nInputPlane*kH*kW x 1

    // Unpack columns back into input:
    col2im(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, fgradInput_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, gradInput_n)
    );

    THCTensor_(free)(state, gradOutput3d);
    THCTensor_(free)(state, fgradInput3d);
    THCTensor_(free)(state, wslice);
    THCTensor_(free)(state, gislice);
    THCTensor_(free)(state, goslice);
  }

  // Free
  THCTensor_(free)(state, gradInput_n);
  THCTensor_(free)(state, fgradInput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize output
  if (batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize3d)(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }

  THCTensor_(transpose)(state, weight, weight, 1, 2);
}

void THNN_(SpatialConvolutionLocal_accGradParameters)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradWeight,
           THCTensor *gradBias,
           THCTensor *finput,
           THCTensor *fgradInput,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           long inputWidth, long inputHeight,
           long outputWidth, long outputHeight,
           real scale)
{
  THCUNN_assertSameGPU_generic(state, 5, input, gradOutput, gradWeight,
                                 gradBias, finput);

  long nInputPlane = THCTensor_(size)(state,gradWeight,2)/(kW*kH);
  long nOutputPlane = THCTensor_(size)(state,gradWeight,1);

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCTensor_(resize4d)(state, input, 1, nInputPlane, inputHeight, inputWidth);
    THCTensor_(resize4d)(state, gradOutput, 1, nOutputPlane, outputHeight, outputWidth);
  }

  // Batch size + input planes
  long batchSize = input->size[0];

  // Helpers
  THCTensor *input_n = THCTensor_(new)(state);
  THCTensor *finput_n = THCTensor_(new)(state);
  THCTensor *gradOutput_n = THCTensor_(new)(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    THCTensor *gradOutput3d, *finput3d;
    THCTensor *gwslice = THCTensor_(new)(state);
    THCTensor *islice = THCTensor_(new)(state);
    THCTensor *goslice = THCTensor_(new)(state);

    // Matrix mulitply per output:
    THCTensor_(select)(state, input_n, input, 0, elt);
    THCTensor_(select)(state, finput_n, finput, 0, elt);
    THCTensor_(select)(state, gradOutput_n, gradOutput, 0, elt);

    gradOutput3d = THCTensor_(newWithStorage3d)(state, gradOutput_n->storage, gradOutput_n->storageOffset,
                                                 outputHeight*outputWidth, 1,
                                                 nOutputPlane, outputHeight*outputWidth,
                                                 1, nOutputPlane*outputHeight*outputWidth);
    finput3d = THCTensor_(newWithStorage3d)(state, finput_n->storage, finput_n->storageOffset,
                                             outputHeight*outputWidth, 1,
                                             1, kW*kH*nInputPlane*outputHeight*outputWidth,
                                             kW*kH*nInputPlane, outputHeight*outputWidth);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCTensor_(data)(state, input_n),
      nInputPlane, inputHeight, inputWidth, kH, kW, padH, padW, dH, dW,
      1, 1, THCTensor_(data)(state, finput_n)
    );

    for (int i = 0; i < outputHeight; i++) {
      for(int j = 0; j < outputWidth; j++) {
        int sliceidx = i * outputWidth + j;
        THCTensor_(select)(state, gwslice, gradWeight, 0, sliceidx);
        THCTensor_(select)(state, goslice, gradOutput3d, 0, sliceidx);
        THCTensor_(select)(state, islice, finput3d, 0, sliceidx);
        THCTensor_(addmm)(state, gwslice, ScalarConvert<int, real>::to(1), gwslice, scale, goslice, islice);
      }
    }
    // gradOutput3d:  oH*oW x nOutputPlane x 1
    // finput3d:      oH*oW x 1 x kW*kH*nInputPlane
    //THCTensor_(baddbmm)(state, gradWeight, 1.0, gradWeight, scale, gradOutput3d, finput3d);
    // gradWeight:    oH*oW x nOutputPlane x kW*kH*nInputPlane

    THCTensor_(cadd)(state, gradBias, gradBias, scale, gradOutput_n);

    THCTensor_(free)(state, gradOutput3d);
    THCTensor_(free)(state, finput3d);
    THCTensor_(free)(state, gwslice);
    THCTensor_(free)(state, goslice);
    THCTensor_(free)(state, islice);
  }

  // Free
  THCTensor_(free)(state, input_n);
  THCTensor_(free)(state, finput_n);
  THCTensor_(free)(state, gradOutput_n);

  // Resize
  if (batch == 0) {
    THCTensor_(resize3d)(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCTensor_(resize3d)(state, input, nInputPlane, inputHeight, inputWidth);
  }
}

#endif