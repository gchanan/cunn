
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialDilatedMaxPooling.cu"
#else

#include "../common.h"

void THNN_(SpatialDilatedMaxPooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCudaLongTensor *indices,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           bool ceil_mode)
{

  THCUNN_assertSameGPU_generic(state, 3, input, output, indices);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  THArgCheck(nInputCols >= kW - padW && nInputRows >= kH - padH, 2, "input image smaller than kernel size");
  THArgCheck(kW/2 >= padW && kH/2 >= padH, 2, "pad should be smaller than half of kernel size");

  if(ceil_mode) {
    nOutputCols = ceil(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
    nOutputRows = ceil(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
  }
  else {
    nOutputCols = floor(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
    nOutputRows = floor(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
  }

if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

if (padW || padH)
  {
    // ensure that the last pooling starts inside the image
    if ((nOutputRows - 1)*dH >= nInputRows + padH)
      --nOutputRows;
    if ((nOutputCols  - 1)*dW >= nInputCols  + padW)
      --nOutputCols;
  }

  input = THCTensor_(newContiguous)(state, input);
  real* input_data = THCTensor_(data)(state, input);

  THCTensor_(resize4d)(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);
  THCUNN_resizeAs_indices(state, indices, output);

  long* indices_data = THCudaLongTensor_data(state, indices);
  real* output_data = THCTensor_(data)(state, output);

  int count = THCTensor_(nElement)(state, output);

  MaxPoolForward<real, accreal> <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, input_data,
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW, output_data, indices_data);
  THCudaCheck(cudaGetLastError());

  if(input->nDimension == 3)
    THCTensor_(resize3d)(state, output, nInputPlane, nOutputRows, nOutputCols);

  THCTensor_(free)(state, input);
}

void THNN_(SpatialDilatedMaxPooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCudaLongTensor *indices,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           bool ceil_mode)
{
  THCUNN_assertSameGPU_generic(state, 4, input, gradOutput, indices, gradInput);
//print("rescuda", rescuda:sub(12,12,37,37,34,35)))
  //real entry1 = THCTensor_(get3d)(state, gradOutput, 12,10,12);
  //real entry2 = THCTensor_(get3d)(state, gradOutput, 12,37,35);(12,12,10,10,12,12))
  //printf("entry (12,10,12) is: %f\n", ScalarConvert<real, accreal>::to(entry1));
  //real entry1 = THCTensor_(get3d)(state, gradOutput, 11,9,11);
  //printf("entry (12,10,12) is: %f\n", ScalarConvert<real, accreal>::to(entry1));
  //printf("entry (12,37,35) is: %f\n", ScalarConvert<real, accreal>::to(entry2));
  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  //entry1 = THCTensor_(get3d)(state, gradOutput, 11,9,11);
  //printf("entry (12,10,12) is: %f\n", ScalarConvert<real, accreal>::to(entry1));

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  if(ceil_mode) {
     nOutputCols = ceil(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
     nOutputRows = ceil(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
   }
   else {
     nOutputCols = floor(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
     nOutputRows = floor(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
   }

  if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);
  printf("COUNTS: %i %i", count, GET_BLOCKS(count));

  MaxPoolBackward<real, accreal> <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
  //MaxPoolBackward<real, accreal> <<< 1, 1, 0, THCState_getCurrentStream(state) >>>
      (count,
      THCTensor_(data)(state, gradOutput),
      THCudaLongTensor_data(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW,
      THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, gradOutput);

  // clean
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif
