
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialDilatedMaxPooling.cu"
#else

#include "../common.h"

void THNN_(SpatialDilatedMaxPooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCTensor *indices,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           bool ceil_mode)
{

  THCUNN_assertSameGPU_generic(state, 3, input, output, indices);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  THArgCheck(nInputCols >= kW - padW && nInputRows >= kH - padH, 2, "input image smaller than kernel size");
  THArgCheck(kW/2 >= padW && kH/2 >= padH, 2, "pad should be smaller than half of kernel size");

  if(ceil_mode) {
    nOutputCols = ScalarConvert<real,long>::to(
      THCNumerics<real>::ceil(ScalarConvert<long,real>::to(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / ScalarConvert<long,real>::to(dW))) + 1;
    nOutputRows = ScalarConvert<real,long>::to(
      THCNumerics<real>::ceil(ScalarConvert<long,real>::to(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / ScalarConvert<long,real>::to(dH))) + 1;
  }
  else {
    nOutputCols = ScalarConvert<real,long>::to(
      THCNumerics<real>::floor(ScalarConvert<long,real>::to(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / ScalarConvert<long,real>::to(dW))) + 1;
    nOutputRows = ScalarConvert<real,long>::to(
      THCNumerics<real>::floor(ScalarConvert<long,real>::to(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / ScalarConvert<long,real>::to(dH))) + 1;
  }

if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

if (padW || padH)
  {
    // ensure that the last pooling starts inside the image
    if ((nOutputRows - 1)*dH >= nInputRows + padH)
      --nOutputRows;
    if ((nOutputCols  - 1)*dW >= nInputCols  + padW)
      --nOutputCols;
  }

  input = THCTensor_(newContiguous)(state, input);
  real* input_data = THCTensor_(data)(state, input);

  THCTensor_(resize4d)(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);
  THCTensor_(resizeAs)(state, indices, output);

  real* indices_data = THCTensor_(data)(state, indices);
  real* output_data = THCTensor_(data)(state, output);

  int count = THCTensor_(nElement)(state, output);

  MaxPoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, input_data,
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW, output_data, indices_data);
  THCudaCheck(cudaGetLastError());

  if(input->nDimension == 3)
    THCTensor_(resize3d)(state, output, nInputPlane, nOutputRows, nOutputCols);

  THCTensor_(free)(state, input);
}

void THNN_(SpatialDilatedMaxPooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *indices,
           int kW, int kH,
           int dW, int dH,
           int padW, int padH,
           int dilationW, int dilationH,
           bool ceil_mode)
{
  THCUNN_assertSameGPU_generic(state, 4, input, gradOutput, indices, gradInput);

  input = THCTensor_(newContiguous)(state, input);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  if(ceil_mode) {
    nOutputCols = ScalarConvert<real,long>::to(
      THCNumerics<real>::ceil(ScalarConvert<long,real>::to(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / ScalarConvert<long,real>::to(dW))) + 1;
    nOutputRows = ScalarConvert<real,long>::to(
      THCNumerics<real>::ceil(ScalarConvert<long,real>::to(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / ScalarConvert<long,real>::to(dH))) + 1;
  }
  else {
    nOutputCols = ScalarConvert<real,long>::to(
      THCNumerics<real>::floor(ScalarConvert<long,real>::to(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / ScalarConvert<long,real>::to(dW))) + 1;
    nOutputRows = ScalarConvert<real,long>::to(
      THCNumerics<real>::floor(ScalarConvert<long,real>::to(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / ScalarConvert<long,real>::to(dH))) + 1;
  }

  if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);

  MaxPoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count,
      THCTensor_(data)(state, gradOutput),
      THCTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW,
      THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, gradOutput);

  // clean
  THCTensor_(free)(state, input);
  THCTensor_(free)(state, gradOutput);
}

#endif
