
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/AbsCriterion.cu"
#else

void THNN_(AbsCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage)
{
  THCUNN_assertSameGPU_generic(state, 2, input, target);

  long size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  real sum = thrust::inner_product(input_data, input_data+size, target_data, ScalarConvert<int, real>::to(0), thrust::plus<real>(), abs_functor<real>());

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, sum);
}

void THNN_(AbsCriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage)
{
  THCUNN_assertSameGPU_generic(state, 3, input, target, gradInput);

  long size = THCTensor_(nElement)(state, input);
  real norm = ScalarConvert<double, real>::to(sizeAverage ? 1./size : 1.);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, abs_updateGradInput_functor<real>(norm));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
