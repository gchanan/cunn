#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "SharedMem.cuh"

template <typename T, typename AccumT>
struct MaxFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT max, T v) const
  {
    return fmaxType(max, v);
  }
};

template<typename T, typename AccumT>
struct SumFloat
{
  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const
  {
    return sum + v;
  }
};

template<typename T, typename AccumT>
struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(T v)
    : max_k(v)
  {}

  __device__ __forceinline__ AccumT operator()(AccumT sum, T v) const
  {
    return sum + THCNumerics<T>::exp(v - max_k);
  }

  const T max_k;
};

template<typename AccumT>
struct NoFinal
{
  __device__ __forceinline__ AccumT operator()(AccumT v) const
  {
    return v;
  }
};

template<typename AccumT>
struct LSMFinal
{
  __device__ __forceinline__ LSMFinal(AccumT m)
    : max_k(m)
  {}

  __device__ __forceinline__ AccumT operator()(AccumT v) const
  {
    return max_k + THCNumerics<AccumT>::log(v);
  }

  const AccumT max_k;
};

template <template<typename, typename> class Reduction, template<typename> class Finalize, typename AccumT>
__device__ __forceinline__ AccumT
blockReduce(AccumT* smem, AccumT val,
            const Reduction<AccumT, AccumT>& r,
            AccumT defaultVal,
            const Finalize<AccumT>& f)
{
  // To avoid RaW races from chaining blockReduce calls together, we
  // need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  AccumT warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  if ((threadIdx.x / 32) == 0) // only threads in warp1 go into this (if)
  {
    int lane = threadIdx.x % 32; // from 0 to 31

    // if less than 1024 threads per block, then only activate the relevant lanes
    if (lane < blockDim.x / 32)
    {
#pragma unroll
      for (int i = 0; i < 32; ++i)
      {
        warpVal = r(warpVal, smem[lane * 32 + i]);
      }

      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumT blockVal = defaultVal;

  if (threadIdx.x == 0)
  {
    for (int i = 0; i < blockDim.x / 32; ++i)
    {
      blockVal = r(blockVal, smem[i]);
    }

    smem[0] = f(blockVal);
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <template<typename, typename> class Reduction, typename AccumT>
__device__ __forceinline__ AccumT
blockReduce(AccumT* smem, AccumT val,
            const Reduction<AccumT, AccumT>& r,
            AccumT defaultVal)
{
  return blockReduce<Reduction, NoFinal, AccumT>(smem, val, r, defaultVal, NoFinal<AccumT>());
}

template <template<typename, typename> class Reduction, int ILP, typename T, typename AccumT>
__device__ __forceinline__ AccumT
ilpReduce(T* data,
          int size,
          const Reduction<T, AccumT>& r,
          AccumT defaultVal)
{
  AccumT threadVal = defaultVal;
  int offset = threadIdx.x;

  int last = size % (ILP * blockDim.x);

  // Body (unroll by ILP times)
  for (; offset < size - last; offset += blockDim.x * ILP)
  {
    T tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      tmp[j] = data[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      threadVal = r(threadVal, tmp[j]);
    }
  }

  // Epilogue
  for (; offset < size; offset += blockDim.x)
  {
    threadVal = r(threadVal, data[offset]);
  }

  return threadVal;
}

template <int ILP, typename T, typename AccumT>
__global__ void
cunn_LogSoftMax_updateOutput_kernel(T *output, T *input, int classes)
{
  SharedMem<AccumT> smem;
  AccumT *buffer = smem.getPointer();
  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * classes;
  output += blockIdx.x * classes;

  // find the max of the batch
  AccumT threadMax = ilpReduce<MaxFloat, ILP, T, AccumT>(
      input, classes, MaxFloat<T, AccumT>(), -THCNumerics<AccumT>::max());
  // find the max over all batches
  AccumT max_k = blockReduce<MaxFloat, AccumT>(
      buffer, threadMax, MaxFloat<AccumT, AccumT>(), -THCNumerics<AccumT>::max());
  T max_k_non_accum = ScalarConvert<AccumT, T>::to(max_k);

  AccumT threadExp = ilpReduce<SumExpFloat, ILP, T, AccumT>(
      input, classes, SumExpFloat<T, AccumT>(max_k_non_accum), 0.0);
  T logsum_k = ScalarConvert<AccumT, T>::to(
      blockReduce<SumFloat, LSMFinal, AccumT>(
          buffer, threadExp, SumFloat<AccumT, AccumT>(), 0.0, LSMFinal<AccumT>(max_k)));

  // Output LSM (hand ILP)
  int offset = threadIdx.x;

  int last = classes % (ILP * blockDim.x);
  for (; offset < classes - last; offset += blockDim.x * ILP)
  {
    T tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      output[offset + j * blockDim.x] = tmp[j] - logsum_k;
    }
  }

  for (; offset < classes; offset += blockDim.x)
  {
    output[offset] = input[offset] - logsum_k;
  }
}

template <int ILP, typename T, typename AccumT>
__global__ void
cunn_LogSoftMax_updateGradInput_kernel(T *gradInput,
                                       T *output,
                                       T *gradOutput,
                                       int classes)
{
  SharedMem<AccumT> smem;
  AccumT *buffer = smem.getPointer();
  gradInput += blockIdx.x * classes;
  output += blockIdx.x * classes;
  gradOutput += blockIdx.x * classes;

  AccumT threadSum = ilpReduce<SumFloat, 4, T, AccumT>(
      gradOutput, classes, SumFloat<T, AccumT>(), 0.0);
  T sum_k = ScalarConvert<AccumT, T>::to(
      blockReduce<SumFloat, AccumT>(
          buffer, threadSum, SumFloat<AccumT, AccumT>(), 0.0));

  // Update gradInput (hand ILP)
  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);
  for (; offset < classes - last; offset += blockDim.x * ILP)
  {
    T tmpGradOutput[ILP];
    T tmpOutput[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      tmpGradOutput[j] = gradOutput[offset + j * blockDim.x];
      tmpOutput[j] = output[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      gradInput[offset + j * blockDim.x] =
        tmpGradOutput[j] - fastExpIfAvail(tmpOutput[j]) * sum_k;
    }
  }

  for (; offset < classes; offset += blockDim.x)
  {
    gradInput[offset] =
      gradOutput[offset] - fastExpIfAvail(output[offset]) * sum_k;
  }
}

#include "generic/LogSoftMax.cu"
#include "THCGenerateFloatTypes.h"
