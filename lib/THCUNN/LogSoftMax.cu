#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "SharedMem.cuh"

template <typename T, typename AccumulatorT>
struct MaxFloat
{
  __device__ __forceinline__ AccumulatorT operator()(AccumulatorT max, T v) const
  {
    return fmaxType(max, v);
  }
};

template<typename T, typename AccumulatorT>
struct SumFloat
{
  __device__ __forceinline__ AccumulatorT operator()(AccumulatorT sum, T v) const
  {
    return sum + v;
  }
};

template<typename T, typename AccumulatorT>
struct SumExpFloat
{
  __device__ __forceinline__ SumExpFloat(T v)
    : max_k(v)
  {}

  __device__ __forceinline__ AccumulatorT operator()(AccumulatorT sum, T v) const
  {
    return sum + THCNumerics<T>::exp(v - max_k);
  }

  const T max_k;
};

template<typename AccumulatorT>
struct NoFinal
{
  __device__ __forceinline__ AccumulatorT operator()(AccumulatorT v) const
  {
    return v;
  }
};

template<typename AccumulatorT>
struct LSMFinal
{
  __device__ __forceinline__ LSMFinal(AccumulatorT m)
    : max_k(m)
  {}

  __device__ __forceinline__ AccumulatorT operator()(AccumulatorT v) const
  {
    return max_k + THCNumerics<AccumulatorT>::exp(v);
  }

  const AccumulatorT max_k;
};

template <template<typename, typename> class Reduction, template<typename> class Finalize, typename AccumulatorT>
__device__ __forceinline__ AccumulatorT
blockReduce(AccumulatorT* smem, AccumulatorT val,
            const Reduction<AccumulatorT, AccumulatorT>& r,
            AccumulatorT defaultVal,
            const Finalize<AccumulatorT>& f)
{
  // To avoid RaW races from chaining blockReduce calls together, we
  // need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  AccumulatorT warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  if ((threadIdx.x / 32) == 0) // only threads in warp1 go into this (if)
  {
    int lane = threadIdx.x % 32; // from 0 to 31

    // if less than 1024 threads per block, then only activate the relevant lanes
    if (lane < blockDim.x / 32)
    {
#pragma unroll
      for (int i = 0; i < 32; ++i)
      {
        warpVal = r(warpVal, smem[lane * 32 + i]);
      }

      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  AccumulatorT blockVal = defaultVal;

  if (threadIdx.x == 0)
  {
    for (int i = 0; i < blockDim.x / 32; ++i)
    {
      blockVal = r(blockVal, smem[i]);
    }

    smem[0] = f(blockVal);
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <template<typename, typename> class Reduction, typename AccumulatorT>
__device__ __forceinline__ AccumulatorT
blockReduce(AccumulatorT* smem, AccumulatorT val,
            const Reduction<AccumulatorT, AccumulatorT>& r,
            AccumulatorT defaultVal)
{
  return blockReduce<Reduction, NoFinal, AccumulatorT>(smem, val, r, defaultVal, NoFinal<AccumulatorT>());
}

template <template<typename, typename> class Reduction, int ILP, typename T, typename AccumulatorT>
__device__ __forceinline__ AccumulatorT
ilpReduce(T* data,
          int size,
          const Reduction<T, AccumulatorT>& r,
          AccumulatorT defaultVal)
{
  AccumulatorT threadVal = defaultVal;
  int offset = threadIdx.x;

  int last = size % (ILP * blockDim.x);

  // Body (unroll by ILP times)
  for (; offset < size - last; offset += blockDim.x * ILP)
  {
    T tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      tmp[j] = data[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      threadVal = r(threadVal, tmp[j]);
    }
  }

  // Epilogue
  for (; offset < size; offset += blockDim.x)
  {
    threadVal = r(threadVal, data[offset]);
  }

  return threadVal;
}

template <int ILP, typename T, typename AccumulatorT>
__global__ void
cunn_LogSoftMax_updateOutput_kernel(T *output, T *input, int classes)
{
  SharedMem<AccumulatorT> smem;
  AccumulatorT *buffer = smem.getPointer();
  // forward pointers to batch[blockIdx.x]
  // each block handles a sample in the mini-batch
  input += blockIdx.x * classes;
  output += blockIdx.x * classes;

  // find the max of the batch
  AccumulatorT threadMax = ilpReduce<MaxFloat, ILP, T, AccumulatorT>(
      input, classes, MaxFloat<T, AccumulatorT>(), -THCNumerics<AccumulatorT>::max());
  // find the max over all batches
  AccumulatorT max_k = blockReduce<MaxFloat, AccumulatorT>(
      buffer, threadMax, MaxFloat<AccumulatorT, AccumulatorT>(), -THCNumerics<AccumulatorT>::max());
  T max_k_non_accum = ScalarConvert<AccumulatorT, T>::to(max_k);

  AccumulatorT threadExp = ilpReduce<SumExpFloat, ILP, T, AccumulatorT>(
      input, classes, SumExpFloat<T, AccumulatorT>(max_k_non_accum), 0.0);
  T logsum_k = ScalarConvert<AccumulatorT, T>::to(
      blockReduce<SumFloat, LSMFinal, AccumulatorT>(
          buffer, threadExp, SumFloat<AccumulatorT, AccumulatorT>(), 0.0, LSMFinal<AccumulatorT>(max_k)));

  // Output LSM (hand ILP)
  int offset = threadIdx.x;

  int last = classes % (ILP * blockDim.x);
  for (; offset < classes - last; offset += blockDim.x * ILP)
  {
    T tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      output[offset + j * blockDim.x] = tmp[j] - logsum_k;
    }
  }

  for (; offset < classes; offset += blockDim.x)
  {
    output[offset] = input[offset] - logsum_k;
  }
}

template <int ILP, typename T, typename AccumulatorT>
__global__ void
cunn_LogSoftMax_updateGradInput_kernel(T *gradInput,
                                       T *output,
                                       T *gradOutput,
                                       int classes)
{
  SharedMem<AccumulatorT> smem;
  AccumulatorT *buffer = smem.getPointer();
  gradInput += blockIdx.x * classes;
  output += blockIdx.x * classes;
  gradOutput += blockIdx.x * classes;

  AccumulatorT threadSum = ilpReduce<SumFloat, 4, T, AccumulatorT>(
      gradOutput, classes, SumFloat<T, AccumulatorT>(), 0.0);
  T sum_k = ScalarConvert<AccumulatorT, T>::to(
      blockReduce<SumFloat, AccumulatorT>(
          buffer, threadSum, SumFloat<AccumulatorT, AccumulatorT>(), 0.0));

  // Update gradInput (hand ILP)
  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);
  for (; offset < classes - last; offset += blockDim.x * ILP)
  {
    T tmpGradOutput[ILP];
    T tmpOutput[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      tmpGradOutput[j] = gradOutput[offset + j * blockDim.x];
      tmpOutput[j] = output[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j)
    {
      gradInput[offset + j * blockDim.x] =
        tmpGradOutput[j] - fastExpIfAvail(tmpOutput[j]) * sum_k;
    }
  }

  for (; offset < classes; offset += blockDim.x)
  {
    gradInput[offset] =
      gradOutput[offset] - fastExpIfAvail(output[offset]) * sum_k;
  }
}

#include "generic/LogSoftMax.cu"
#include "THCGenerateFloatTypes.h"
